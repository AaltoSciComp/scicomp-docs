
#include <hip/hip_runtime.h>
#include "stdio.h"

__global__ void cuda_hello(int* a){
        // blockIdx has values between 0 and 4
        printf("Hello World from GPU a[%d]=%d \n", blockIdx.x, a[blockIdx.x]);
}

int main(void) {
        int* d_a;

        // Allocates an array of 5 integers
        hipMalloc(&d_a, 5*sizeof(int));

        // Runs 5 instances of kernel cuda_hello in parallel
        cuda_hello<<<5, 1>>>(d_a); 

        // This is needed for the printf in the kernel to display
        hipDeviceSynchronize();

        printf("Hello from outside GPU\n");
        return 0;
}
